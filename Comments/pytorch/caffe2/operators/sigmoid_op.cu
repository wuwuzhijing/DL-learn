#include "hip/hip_runtime.h"
#include "caffe2/operators/sigmoid_op.h"

#include <algorithm>
#include <functional>

#include "caffe2/core/context_gpu.h"

namespace caffe2 {

namespace {

template <typename T>
__global__ void SigmoidKernel(const int N, const T* X, T* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
#if __CUDA_ARCH__ >= 350
    Y[i] = T(1) / (T(1) + exp(-__ldg(X + i)));
#else
    Y[i] = T(1) / (T(1) + exp(-X[i]));
#endif
  }
}

template <typename T>
__global__ void
SigmoidGradientKernel(const int N, const T* dY, const T* Y, T* dX) {
  CUDA_1D_KERNEL_LOOP(i, N) {
#if __CUDA_ARCH__ >= 350
    dX[i] = __ldg(dY + i) * __ldg(Y + i) * (T(1) - __ldg(Y + i));
#else
    dX[i] = dY[i] * Y[i] * (T(1) - Y[i]);
#endif
  }
}

} // namespace

template <>
template <typename T>
bool SigmoidFunctor<HIPContext>::
operator()(const int N, const T* X, T* Y, HIPContext* context) const {
  SigmoidKernel<T>
      <<<CAFFE_GET_BLOCKS(N),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(N, X, Y);
  return true;
}

template <>
template <typename T>
bool SigmoidGradientFunctor<HIPContext>::Forward(
    const std::vector<int>& Y_dims,
    const std::vector<int>& /* dY_dims */,
    const T* Y,
    const T* dY,
    T* dX,
    HIPContext* context) const {
  const int size = std::accumulate(
      Y_dims.cbegin(), Y_dims.cend(), 1, std::multiplies<int>());
  SigmoidGradientKernel<T>
      <<<CAFFE_GET_BLOCKS(size),
         CAFFE_CUDA_NUM_THREADS,
         0,
         context->hip_stream()>>>(size, dY, Y, dX);
  return true;
}

REGISTER_HIP_OPERATOR(
    Sigmoid,
    UnaryElementwiseOp<
        TensorTypes<float>,
        HIPContext,
        SigmoidFunctor<HIPContext>>);
REGISTER_HIP_OPERATOR(
    SigmoidGradient,
    BinaryElementwiseOp<
        TensorTypes<float>,
        HIPContext,
        SigmoidGradientFunctor<HIPContext>>);

} // namespace caffe2
